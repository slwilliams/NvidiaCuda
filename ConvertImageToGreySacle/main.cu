#include "hip/hip_runtime.h"
#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
  
  uchar4 rgba = rgbaImage[blockIdx.x * numCols + blockIdx.y];
  float channelSum = .299f * rgba.x + .587f * rgba.y + .114f * rgba.z;
  greyImage[blockIdx.x * numCols + blockIdx.y] = channelSum;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(1, 1, 1);  //TODO
  const dim3 gridSize(numRows, numCols, 1);  //TODO
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}